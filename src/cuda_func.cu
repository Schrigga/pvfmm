
#include <hip/hip_runtime.h>
#include "stdint.h"

#define DEFAULT_NUM_THREAD 256

/* Case: double */
__global__ void in_perm_k (
  void *precomp_data,
  size_t *input_perm,
  void *input_data,
  void *buff_in,
  size_t interac_indx,
  size_t M_dim0,
  size_t vec_cnt )
{
  /* 1-dim thread Id. */
  int tid = blockIdx.x*blockDim.x + threadIdx.x;

  /* Convert to ptr. */
/*
  int *perm = (int*) (precomp_data + input_perm[(interac_indx + tid)*4 + 0]);
  double *scal = (double*) (precomp_data + input_perm[(interac_indx + tid)*4 + 1]);
  double *v_in = (double*) (input_data[0] + input_perm[(interac_indx + tid)*4 + 3]);
  double *v_out = (double*) (buff_in + input_perm[(interac_indx + tid)*4 + 2]);
*/
  if (tid < vec_cnt) {
    /* PRAM Model: assuming as many threads as we need. */
    //for (int j = 0; j < M_dim0; j++) v_out[j] = v_in[perm[j]]*scal[j];
  }
}

__global__ void out_perm_k (
  double *scaling,
  void *precomp_data,
  size_t *output_perm,
  void *output_data,
  void *buff_out,
  size_t interac_indx,
  size_t M_dim1,
  size_t vec_cnt )
{
  /* 1-dim thread Id. */
  int tid = blockIdx.x*blockDim.x + threadIdx.x;

  /* Specifing range. */
  int a = tid;
  int b = tid + 1;

  if (tid > 0 && a < vec_cnt) { // Find 'a' independent of other threads.
    size_t out_ptr = output_perm[(interac_indx + a)*4 + 3];
    if (tid > 0) while(a < vec_cnt && out_ptr == output_perm[(interac_indx+a)*4 + 3]) a++;
  }
  if (tid < vec_cnt - 1 && b < vec_cnt) { // Find 'b' independent of other threads.
    size_t out_ptr = output_perm[(interac_indx + b)*4 + 3];
    if (tid < vec_cnt-1) while(b < vec_cnt && out_ptr == output_perm[(interac_indx+b)*4 + 3]) b++;
  }

  if (tid < vec_cnt) {
    /* PRAM Model: assuming as many threads as we need. */
    for(int i = a; i < b; i++) { // Compute permutations.
/*
      double scaling_factor = scaling[interac_indx + i];
      int *perm = (int*) (precomp_data + output_perm[(interac_indx + i)*4 + 0]);
      double *scal = (double*) (precomp_data + output_perm[(interac_indx + i)*4 + 1]);
      double *v_in = (double*) (buff_out + output_perm[(interac_indx + i)*4 + 3]);
      double *v_out = (double*) (output_data + output_perm[(interac_indx + i)*4 + 2]);
      for (int j = 0; j < M_dim1; j++) v_out[j] += v_in[perm[j]]*scal[j]*scaling_factor;
*/
    }
  }
}

extern "C" 
void in_perm_d (
  uintptr_t precomp_data,
  uintptr_t input_perm,
  uintptr_t input_data,
  uintptr_t buff_in,
  size_t interac_indx,
  size_t M_dim0,
  size_t vec_cnt,
  hipStream_t *stream )
{
  int n_thread, n_block;
  n_thread = DEFAULT_NUM_THREAD;
  n_block = vec_cnt/n_thread;
  void *precomp_data_d = (void *) precomp_data;
  size_t *input_perm_d = (size_t *) input_perm;
  void *input_data_d = (void *) input_data;
  void *buff_in_d = (void *) buff_in;
  in_perm_k<<<n_thread, n_block, 0, *stream>>>(precomp_data_d, input_perm_d, input_data_d,
    buff_in_d, interac_indx, M_dim0, vec_cnt);
}

extern "C"
void out_perm_d (
  uintptr_t scaling,
  uintptr_t precomp_data,
  uintptr_t output_perm,
  uintptr_t output_data,
  uintptr_t buff_out,
  size_t interac_indx,
  size_t M_dim0,
  size_t vec_cnt,
  hipStream_t *stream )
{
  int n_thread, n_block;
  n_thread = DEFAULT_NUM_THREAD;
  n_block = vec_cnt/n_thread;
  double *scaling_d = (double *) scaling;
  void *precomp_data_d = (void *) precomp_data;
  size_t *output_perm_d = (size_t *) output_perm;
  void *output_data_d = (void *) output_data;
  void *buff_out_d = (void *) buff_out;
  out_perm_k<<<n_thread, n_block, 0, *stream>>>(scaling_d, precomp_data_d, output_perm_d, 
    output_data_d, buff_out_d, interac_indx, M_dim0, vec_cnt);
}
