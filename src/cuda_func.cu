
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <cassert>

__global__
void  in_perm_k(char* precomp_data, double*  input_data, char* buff_in , size_t*  input_perm, size_t vec_cnt, size_t M_dim0){
  extern __shared__ double s[];

  /* Specifing range. */
  int a = ( blockIdx.x     *vec_cnt)/gridDim.x;
  int b = ((blockIdx.x + 1)*vec_cnt)/gridDim.x;

  for(int i = a; i < b; i++) { // Compute permutations.
    const size_t* perm= (size_t*) (precomp_data + input_perm[i*4+0]);
    const double* scal= (double*) (precomp_data + input_perm[i*4+1]);
    const double*v_in = (double*) (input_data   + input_perm[i*4+3]);
    double*      v_out= (double*) (buff_in      + input_perm[i*4+2]);
    for (size_t j = threadIdx.x; j < M_dim0; j+=blockDim.x) s[j] = v_in[j];
    __syncthreads();
    for (size_t j = threadIdx.x; j < M_dim0; j+=blockDim.x) v_out[j] = s[perm[j]]*scal[j];
    __syncthreads();
  }
};

__global__
void out_perm_k(char* precomp_data, double* output_data, char* buff_out, size_t* output_perm, size_t vec_cnt, size_t M_dim1){
  extern __shared__ double s[];
  for (size_t j = threadIdx.x; j < M_dim1; j+=blockDim.x) s[j] = 0;

  /* Specifing range. */
  int a = ( blockIdx.x     *vec_cnt)/gridDim.x;
  int b = ((blockIdx.x + 1)*vec_cnt)/gridDim.x;

  if (blockIdx.x > 0             && a < vec_cnt) { // Find 'a' independent of other threads.
    size_t out_ptr = output_perm[a*4+3];
    if (blockIdx.x >             0) while (a < vec_cnt && out_ptr == output_perm[a*4+3]) a++;
  }
  if (blockIdx.x < gridDim.x - 1 && b < vec_cnt) { // Find 'b' independent of other threads.
    size_t out_ptr = output_perm[b*4+3];
    if (blockIdx.x < gridDim.x - 1) while (b < vec_cnt && out_ptr == output_perm[b*4+3]) b++;
  }

  for(int i = a; i < b; i++) { // Compute permutations.
    size_t  *perm = (size_t*) (precomp_data + output_perm[i*4+0]);
    double  *scal = (double*) (precomp_data + output_perm[i*4+1]);
    double *v_in  = (double*) (buff_out     + output_perm[i*4+2]);
    double *v_out = (double*) (output_data  + output_perm[i*4+3]);
    for(size_t j = threadIdx.x; j<M_dim1; j+=blockDim.x){
      s[j] += v_in[perm[j]]*scal[j];
    }
    if(output_perm[i*4+3]!=output_perm[(i+1)*4+3])
    for(size_t j = threadIdx.x; j<M_dim1; j+=blockDim.x){
      v_out[j]+=s[j];
      s[j] = 0;
    }
  }
};

extern "C" {

  void  in_perm_d(char* precomp_data, double*  input_data, char* buff_in , size_t*  input_perm, size_t vec_cnt, size_t M_dim0, hipStream_t *stream){
    if (vec_cnt == 0) return;
    in_perm_k <<<1024, 256, M_dim0*sizeof(double), *stream>>>(precomp_data,  input_data, buff_in ,  input_perm, vec_cnt, M_dim0);
    hipError_t error = hipGetLastError();
    assert(error == hipSuccess);
  };

  void out_perm_d(char* precomp_data, double* output_data, char* buff_out, size_t* output_perm, size_t vec_cnt, size_t M_dim1, hipStream_t *stream){
    if (vec_cnt == 0) return;
    out_perm_k<<<1024, 256, M_dim1*sizeof(double), *stream>>>(precomp_data, output_data, buff_out, output_perm, vec_cnt, M_dim1);
    hipError_t error = hipGetLastError();
    assert(error == hipSuccess);
  };

}
