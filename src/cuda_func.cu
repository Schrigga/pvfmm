
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#define DEFAULT_NUM_THREAD 256

/* Case: double */
__global__ void in_perm_k (
  char *precomp_data,
  size_t *input_perm,
  char *input_data,
  char *buff_in,
  size_t interac_indx,
  size_t M_dim0,
  size_t vec_cnt )
{
  /* 1-dim thread Id. */
  int tid = blockIdx.x*blockDim.x + threadIdx.x;

  size_t s_pdata = ((size_t *) precomp_data)[input_perm[(interac_indx + tid)*4 + 0]/8];
  size_t s_pdata1 = ((size_t *) precomp_data)[input_perm[(interac_indx + tid)*4 + 0]/8 + 1];
  size_t s_pdata2 = ((size_t *) precomp_data)[input_perm[(interac_indx + tid)*4 + 0]/8 + 2];
  double d_pdata = ((double *) precomp_data)[input_perm[(interac_indx + tid)*4 + 1]/8];
  double d_pdata1 = ((double *) precomp_data)[input_perm[(interac_indx + tid)*4 + 1]/8 + 1];
  double d_pdata2 = ((double *) precomp_data)[input_perm[(interac_indx + tid)*4 + 1]/8 + 2];

  if (tid < vec_cnt) {
    /* Convert to ptr. */
    size_t *perm  = (size_t *) (precomp_data + input_perm[(interac_indx + tid)*4 + 0]);
    double *scal  = (double *) (precomp_data + input_perm[(interac_indx + tid)*4 + 1]);
    double *v_in  = (double *) (input_data   + input_perm[(interac_indx + tid)*4 + 3]);
    double *v_out = (double *) (buff_in      + input_perm[(interac_indx + tid)*4 + 2]);

    /* PRAM Model: assuming as many threads as we need. */
    for (size_t j = 0; j < M_dim0; j++) {
/*
      size_t perm_tmp = perm[j];
      double scal_tmp = scal[j];
      double v_in_tmp = v_in[perm_tmp];
      v_out[j] = 0.0;
      v_out[j] = v_in_tmp*scal_tmp;
*/
      v_out[j] = v_in[perm[j]]*scal[j];
    }
  }
}

__global__ void out_perm_k (
  double *scaling,
  char *precomp_data,
  size_t *output_perm,
  char *output_data,
  char *buff_out,
  size_t interac_indx,
  size_t M_dim1,
  size_t vec_cnt )
{
  /* 1-dim thread Id. */
  int tid = blockIdx.x*blockDim.x + threadIdx.x;

  /* Specifing range. */
  int a = (tid*vec_cnt)/vec_cnt;
  int b = ((tid + 1)*vec_cnt)/vec_cnt;

  if (tid > 0 && a < vec_cnt) { // Find 'a' independent of other threads.
    size_t out_ptr = output_perm[(interac_indx + a)*4 + 3];
    if (tid > 0) while (a < vec_cnt && out_ptr == output_perm[(interac_indx + a)*4 + 3]) a++;
  }
  if (tid < vec_cnt - 1 &&  - 1 && b < vec_cnt) { // Find 'b' independent of other threads.
    size_t out_ptr = output_perm[(interac_indx + b)*4 + 3];
    if (tid < vec_cnt - 1) while (b < vec_cnt && out_ptr == output_perm[(interac_indx+b)*4 + 3]) b++;
  }

  if (tid < vec_cnt) {
    /* PRAM Model: assuming as many threads as we need. */
    for(int i = a; i < b; i++) { // Compute permutations.
      double scaling_factor = scaling[interac_indx + i];
      size_t *perm = (size_t*) (precomp_data + output_perm[(interac_indx + i)*4 + 0]);
      double *scal = (double*) (precomp_data + output_perm[(interac_indx + i)*4 + 1]);
      double *v_in = (double*) (buff_out + output_perm[(interac_indx + i)*4 + 3]);
      double *v_out = (double*) (output_data + output_perm[(interac_indx + i)*4 + 2]);
      for (int j = 0; j < M_dim1; j++) v_out[j] += v_in[perm[j]]*scal[j]*scaling_factor;
    }
  }
}

extern "C" { 
void test_d (uintptr_t precomp_data, uintptr_t input_perm, uintptr_t input_data, uintptr_t buff_in, 
  int interac_indx, hipStream_t *stream) {};

void in_perm_d (
/*
  uintptr_t precomp_data,
  uintptr_t input_perm,
  uintptr_t input_data,
  uintptr_t buff_in,
*/
  char *precomp_data,
  size_t *input_perm,
  char *input_data,
  char * buff_in,
  size_t interac_indx,
  size_t M_dim0,
  size_t vec_cnt, 
  hipStream_t *stream )
{
  int n_thread, n_block;
  n_thread = DEFAULT_NUM_THREAD;
  n_block = vec_cnt/n_thread + 1;
/*
  char *precomp_data_d = (char *) precomp_data;
  char *input_perm_d = (char *) input_perm;
  char *input_data_d = (char *) input_data;
  char *buff_in_d = (char *) buff_in;
*/
  /*
  in_perm_k<<<n_thread, n_block, 0, *stream>>>(precomp_data, input_perm, input_data, buff_in, 
    interac_indx, M_dim0, vec_cnt);
*/
  printf("vec_cnt: %d, M_dim0: %d\n", (int) vec_cnt, (int) M_dim0);
  in_perm_k<<<n_block, n_thread>>>(precomp_data, input_perm, input_data, buff_in, 
    interac_indx, M_dim0, vec_cnt);
};

void out_perm_d (
  double *scaling,
  char *precomp_data,
  size_t *output_perm,
  char *output_data,
  char *buff_out,
  size_t interac_indx,
  size_t M_dim0,
  size_t vec_cnt,
  hipStream_t *stream )
{
  int n_thread, n_block;
  n_thread = DEFAULT_NUM_THREAD;
  n_block = vec_cnt/n_thread + 1;
/*
  out_perm_k<<<n_block, n_thread, 0, *stream>>>(scaling, precomp_data, output_perm, output_data, buff_out, 
    interac_indx, M_dim0, vec_cnt);
*/
  out_perm_k<<<n_block, n_thread, 0, *stream>>>(scaling, precomp_data, output_perm, output_data, buff_out, 
    interac_indx, M_dim0, vec_cnt);
};

}
