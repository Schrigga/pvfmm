
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

extern "C" {

void* host_malloc_cuda(size_t size){
  void* p;
  hipError_t error = hipHostAlloc(&p, size, hipHostMallocPortable);
  if (error != hipSuccess) fprintf(stderr,"CUDA Error: %s \n", hipGetErrorString(error));
  assert(error == hipSuccess);
  return p;
}

void host_free_cuda(void* p){
  hipError_t error = hipHostFree(p);
  if (error != hipSuccess) fprintf(stderr,"CUDA Error: %s \n", hipGetErrorString(error));
  assert(error == hipSuccess);
}

}
